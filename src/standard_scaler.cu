#include "hip/hip_runtime.h"
#include "Bench.h"
#include "rapidcsv.h"
#include <cmath>
#include <cstddef>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <limits>
#include <memory>
#include <string>
#include <vector>

static void HandleError(hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

void __global__ standard_scaler(double *xv, double std, double mean, int count) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  if (index < count) {
    xv[index] = (xv[index] - mean) / std;
  }
}

double mean(std::vector<double> xv) {
    double sum = 0;
    for (auto x : xv) {
        sum += x;
    }

    return sum / xv.size();
}

double stdf(std::vector<double> xv, double mean_t) {
    double sum = 0;
    for (auto x : xv) {
        sum += pow(x - mean_t, 2);
    }

    return sqrt(sum / xv.size());
}

void spit_csv(std::string filename, std::vector<std::vector<double>> ds, std::vector<std::string>cnames)
{
  std::ofstream out;
  out.open(filename);

  for (auto name : cnames) {
    out << name << ",";
  }

  out << "\n";

  for (int i = 0; i < ds[0].size(); ++i) {
    for (int j = 0; j < cnames.size(); ++j) {
      out << ds[j][i] << ((j == cnames.size() - 1) ? "\n" : ",");
    }
  }

  out.close();
}

int main(int argc, char *argv[]) {
  /**
   * Initialize doc & file path
   */
  std::string file_path = std::string(argv[1]);
  rapidcsv::Document doc(file_path);
  int THREADS = std::stoi(argv[2]);
  int BLOCKS = std::stoi(argv[3]);
  int TB_SWITCH = std::stoi(argv[4]);

  /**
   * Read CSV
   */
  std::vector<double> R = doc.GetColumn<double>("R");
  std::vector<double> G = doc.GetColumn<double>("G");
  std::vector<double> B = doc.GetColumn<double>("B");
  int count = R.size();
  int size = count * sizeof(double);

  /**
   * Calc MEAN and std for columns
   */
   double MEAN_R = mean(R);
   double STD_R = stdf(R, MEAN_R);

   double MEAN_G = mean(G);
   double STD_G = stdf(G, MEAN_G);

   double MEAN_B = mean(B);
   double STD_B = stdf(B, MEAN_B);

  std::cout << "BLOCKS: " << BLOCKS
            << "\nTHREADS: " << THREADS
            << "\nCOUNT: " << THREADS * BLOCKS
            << "\n\n---------------------------------------\n";

  /**
   * Create global bencher
   */
  std::unique_ptr<Bench> bencher = std::make_unique<Bench>(Bench());

  for (int i = 0; i < 30; ++i) {
    int op_id = bencher->add_op(std::to_string(i));

    /**
     * Create device vars
     */
    double *d_R, *d_G, *d_B;

    /**
     * Alloc device memory
     */
    HANDLE_ERROR(hipMalloc((void **)&d_R, size));
    HANDLE_ERROR(hipMalloc((void **)&d_G, size));
    HANDLE_ERROR(hipMalloc((void **)&d_B, size));

    /**
     * Copy vars to device memory
     */
    HANDLE_ERROR(hipMemcpy(d_R, &R[0], size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_G, &G[0], size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_B, &B[0], size, hipMemcpyHostToDevice));

    /**
     * Algorithm
     */
    standard_scaler<<<BLOCKS, THREADS>>>(d_R, STD_R, MEAN_R, count);
    standard_scaler<<<BLOCKS, THREADS>>>(d_G, STD_G, MEAN_G, count);
    standard_scaler<<<BLOCKS, THREADS>>>(d_B, STD_B, MEAN_B, count);

    /**
     * Copy modified data to host
     */
    HANDLE_ERROR(hipMemcpy(&R[0], d_R, size, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(&G[0], d_G, size, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(&B[0], d_B, size, hipMemcpyDeviceToHost));

    bencher->end_op(op_id);

    std::cout << "\nRUN: " << i + 1 << ", TIME: " << bencher->op_timestamp(op_id) << "ms";
    /*
     * Deallocate memory
     */
    HANDLE_ERROR(hipFree(d_R));
    HANDLE_ERROR(hipFree(d_G));
    HANDLE_ERROR(hipFree(d_B));
  }

  auto output = std::vector<std::vector<double>>{
    R,
    G,
    B,
    doc.GetColumn<double>("SKIN")
  };

  spit_csv("standard_scaler-skin.csv", output, std::vector<std::string>{"R", "G", "B", "SKIN"});
  bencher->csv_output((TB_SWITCH == 1 ? "T" : "B") + std::string("_standard_scaler") +
                      (TB_SWITCH == 1 ? std ::to_string(THREADS) : std::to_string(BLOCKS)));

  return 0;
}
